#include "hip/hip_runtime.h"
#include <iostream>
#include "calcul.hxx"
#include "timer.hxx"
#include "cuda_check.cuh"

__global__ void vecInit(double *a, double *b, int n)
{
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  double x;
  
  if (id >= n) return;

  x = 1.0 * id;
  a[id] = sin(x)*sin(x);
  b[id] = cos(x)*cos(x);
}

__global__ void vecAdd(double *c, double *a, double *b, int n)
{
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  
  if (id < n)
    c[id] = a[id] + b[id];
}


Calcul_GPU::Calcul_GPU(std::size_t n) : m_n(n)
{
  Timer T1; T1.start();
  
  std::size_t bytes = m_n*sizeof(double);
  CUDA_CHECK_OP(hipMalloc(&d_u, bytes));
  CUDA_CHECK_OP(hipMalloc(&d_v, bytes));
  CUDA_CHECK_OP(hipMalloc(&d_w, bytes));
    
  T1.stop();
  std::cerr << "\t\ttemps init 1 : " << T1.elapsed() << std::endl;
  Timer T2; T2.start();
  
  blockSize = 512;
  gridSize = (int)ceil((double)n/blockSize);
  
  vecInit<<<gridSize, blockSize>>>(d_u, d_v, n);
  CUDA_CHECK_KERNEL();

  T2.stop();
  std::cerr << "\t\ttemps init 2 : " << T2.elapsed() << std::endl;
}

Calcul_GPU::~Calcul_GPU()
{
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
}

void Calcul_GPU::addition()
{
  Timer T; T.start();
  
  vecAdd<<<gridSize, blockSize>>>(d_w, d_u, d_v, m_n);
  hipDeviceSynchronize();
  T.stop();
  std::cerr << "\t\ttemps add.   : " << T.elapsed() << std::endl;
}

double Calcul_GPU::verification()
{
  Timer T1, T2;

  T1.start();
  
  std::size_t bytes = m_n*sizeof(double);
  std::vector<double> w(m_n);
  hipMemcpy(w.data(), d_w, bytes, hipMemcpyDeviceToHost);

  T1.stop();
  T2.start();
  double s = 0;
  std::size_t i;
  for (i=0; i<m_n; i++)
    s = s + w[i];
  
  s = s/m_n - 1.0;
  
  T2.stop();
  std::cerr << "\t\ttemps verif1 : " << T1.elapsed() << std::endl;
  std::cerr << "\t\ttemps verif2 : " << T2.elapsed() << std::endl;

  return s;
}

