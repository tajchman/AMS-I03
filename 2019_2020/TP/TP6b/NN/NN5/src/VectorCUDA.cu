#include "MatrixVectorCUDA.cuh"

VectorCUDA::VectorCUDA(size_t n) : m_n(n)
{
  hipMalloc(&(c), n*sizeof(double));
}

VectorCUDA::~VectorCUDA()
{
  hipFree(c);
}

MatrixCUDA::MatrixCUDA(size_t n, size_t m) : m_n(n), m_m(m)
{
  hipMalloc(&(c), n*m*sizeof(double));
}

VectorCUDA::~VectorCUDA()
{
  hipFree(c);
}

