#include "timerGPU.h"
#include <hip/hip_runtime.h>
#include <string>

struct _TimerGPU {
  hipEvent_t m_startEvent, m_stopEvent;
};
  
TimerGPU::TimerGPU(const char *s) : m_running(false), m_elapsed(0.0), m_t(new _TimerGPU) {
  hipEventCreate(&(m_t->m_startEvent));
  hipEventCreate(&(m_t->m_stopEvent));
}

TimerGPU::~TimerGPU() {
  hipEventDestroy(m_t->m_startEvent);
  hipEventDestroy(m_t->m_stopEvent);
  delete m_t;
}

void TimerGPU::start() {
  if (not m_running) {
    hipEventRecord(m_t->m_startEvent,0);
    m_running = true;
  }
}

void TimerGPU::stop() {
  if (m_running) {
    float ms;
    
    hipEventRecord(m_t->m_stopEvent,0);
    hipEventSynchronize(m_t->m_stopEvent);
    hipEventElapsedTime(&ms, m_t->m_startEvent, m_t->m_stopEvent);
    
    m_elapsed += ms;
    m_running = false;
  }
}
  
double TimerGPU::elapsed() { return m_elapsed * 0.001; }
  
