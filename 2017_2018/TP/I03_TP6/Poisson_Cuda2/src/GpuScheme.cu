#include "hip/hip_runtime.h"
#include "GpuScheme.hxx"
#include <math.h>
#include <iostream>
#include "CpuValues.hxx"
#include "hip/hip_runtime.h"
#include "ErrCheck.h"

#define BLOCK_SIZE 512

GpuScheme::GpuScheme(const GpuParameters *P) : AbstractScheme(P),
  host_out(NULL), dev_out(NULL), numBlocks(0), m_w(P)
{

  codeName = "Poisson_GPU";
  deviceName = "GPU";

  m_u = new GpuValues(P);
  m_v = new GpuValues(P);
}

void GpuScheme::initialize()
{
  m_u->init();
  m_v->init();
  m_w.init();

  numBlocks = m_u->n_1D() / (BLOCK_SIZE<<1);
  if (m_u->n_1D() % (BLOCK_SIZE<<1))
    numBlocks++;

  host_out = (double*) malloc(numBlocks * sizeof(double));
  CHECK_CUDA(hipMalloc(&dev_out, numBlocks*sizeof(double)));

}

GpuScheme::~GpuScheme()
{
	delete m_u;
	delete m_v;
	delete host_out;
	hipFree(dev_out);
}

__global__ void
gpu_iteration(const double *u, double *v, double lambda, int nx, int ny, int nz)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x ;
  const int j = blockIdx.y * blockDim.y + threadIdx.y ;
  const int k = blockIdx.z * blockDim.z + threadIdx.z ;

  int i_j_k  = i + j*nx + k*nx*ny;

  int im_j_k = nx > 2 ? i+1 + j*nx + k*nx*ny : i_j_k;
  int ip_j_k = nx > 2 ? i-1 + j*nx + k*nx*ny : i_j_k;

  int i_jm_k = ny > 2 ? i + (j-1)*nx + k*nx*ny : i_j_k;
  int i_jp_k = ny > 2 ? i + (j+1)*nx + k*nx*ny : i_j_k;

  int i_j_km = nz > 2 ? i + j*nx + (k-1)*nx*ny : i_j_k;
  int i_j_kp = nz > 2 ? i + j*nx + (k+1)*nx*ny : i_j_k;

  if (i>0 && i<nx-1 && j>0 && j<ny-1 && k>0 && k<nz-1) {
    v[i_j_k] = u[i_j_k] - lambda *
      (6 * u[i_j_k] - u[ip_j_k] - u[im_j_k]
       - u[i_jp_k] - u[i_jm_k]
       - u[i_j_kp] - u[i_j_km]);
  }
}

__global__ void
gpu_zero(double *u, int n)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x ;

  if (i<n)
    u[i] = 0.0;
}

__global__  void
gpu_norm(const double * input1, const double * input2, double * output, int len)
{
  __shared__ double partialSum[2*BLOCK_SIZE];
  int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int t = threadIdx.x;
  unsigned int start = 2*blockIdx.x*blockDim.x;

  partialSum[t] =	((start + t) < len)
    ? (input1[start + t] - input2[start + t])
    : 0.0;

  partialSum[blockDim.x + t] = ((start + blockDim.x + t) < len)
    ? (input1[start + blockDim.x + t] - input2[start + blockDim.x + t])
    : 0.0;

  for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
    {
      __syncthreads();
      if (t < stride)
	partialSum[t] += partialSum[t + stride];
    }
  __syncthreads();

  if (t == 0 && (globalThreadId*2) < len)
    output[blockIdx.x] = partialSum[t];
}

bool GpuScheme::iteration()
{
  const GpuParameters * p = dynamic_cast<const GpuParameters *>(m_P);
  const sGPU * g = p->GpuInfo;
  size_t n = m_n[0] * m_n[1] * m_n[2];

  gpu_iteration<<<g->dimGrid, g->dimBlock>>>
    (m_u->data(), m_v->data(), m_lambda, m_n[0], m_n[1], m_n[2]);
  
  gpu_norm<<<numBlocks, BLOCK_SIZE>>>(m_u->data(), m_v->data(), dev_out, n);
  
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(host_out, dev_out,
			numBlocks * sizeof(double),
			hipMemcpyDeviceToHost));

  m_duv_max = 0.0;
  for (int i = 0; i < numBlocks; i++)
    m_duv_max += host_out[i];
  
  return true;
}

const AbstractValues & GpuScheme::getOutput()
{
  size_t n = m_n[0] * m_n[1] * m_n[2] * sizeof(double);
  CHECK_CUDA(hipMemcpy(m_w.data(), m_u->data(), n, hipMemcpyDeviceToHost));
  return m_w;
}

void GpuScheme::setInput(const AbstractValues & u)
{
  size_t n = m_n[0] * m_n[1] * m_n[2] * sizeof(double);

  const CpuValues * u1 = dynamic_cast<const CpuValues *>(&u);
  if (u1) {
    CHECK_CUDA(hipMemcpy(m_u->data(), u1->data(), n, hipMemcpyHostToDevice));
    return;
  }
}

