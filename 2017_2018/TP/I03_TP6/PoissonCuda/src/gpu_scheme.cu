#include "hip/hip_runtime.h"
#include "GPU.hxx"
#include "gpu_scheme.hxx"
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

struct sGPU {

	hipDevice_t device;
	hipCtx_t context;
	dim3 dimBlock, dimGrid;
};

GPUScheme::GPUScheme()
{
	deviceName = "GPU";
	m_GPU = new sGPU;

	int deviceCount;
	CHECK_CUDA_RESULT(hipInit(0));
	CHECK_CUDA_RESULT(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		std::cerr << "GPU : no CUDA device found" << std::endl;
		exit(1);
	}
	else {
		std::cerr << "GPU : " << deviceCount << " CUDA device";
		if (deviceCount > 1) std::cerr << "s";
		std::cerr << " found\n" << std::endl;
	}
	CHECK_CUDA_RESULT(hipDeviceGet(&(m_GPU->device), 0));
	CHECK_CUDA_RESULT(hipCtxCreate(&(m_GPU->context), 0, m_GPU->device));

	#define BLOCK_SIZE_X 4
    #define BLOCK_SIZE_Y 4
    #define BLOCK_SIZE_Z 4

    m_GPU->dimBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    m_GPU->dimGrid  = dim3( ceil(float(m_n[0])/float(m_GPU->dimBlock.x)),
                        ceil(float(m_n[1])/float(m_GPU->dimBlock.y)),
                        ceil(float(m_n[2])/float(m_GPU->dimBlock.z)));
}

GPUScheme::~GPUScheme()
{
   CHECK_CUDA_RESULT(hipCtxDestroy(m_GPU->context));
}

__global__ void
gpu_iteration(const double *u, double *v, double lambda, int nx, int ny, int nz)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x ;
	const int j = blockIdx.y * blockDim.y + threadIdx.y ;
	const int k = blockIdx.z * blockDim.z + threadIdx.z ;

	int i_j_k  = i + j*nx + k*nx*ny;

	int im_j_k = nx > 2 ? i+1 + j*nx + k*nx*ny : i_j_k;
	int ip_j_k = nx > 2 ? i-1 + j*nx + k*nx*ny : i_j_k;

	int i_jm_k = ny > 2 ? i + (j-1)*nx + k*nx*ny : i_j_k;
	int i_jp_k = ny > 2 ? i + (j+1)*nx + k*nx*ny : i_j_k;

	int i_j_km = nz > 2 ? i + j*nx + (k-1)*nx*ny : i_j_k;
	int i_j_kp = nz > 2 ? i + j*nx + (k+1)*nx*ny : i_j_k;

    if (i>0 && i<nx && j>0 && j<ny && k>0 && k<nz)
    	v[i_j_k] = u[i_j_k] - lambda *
    	    (6 * u[i_j_k] - u[ip_j_k] - u[im_j_k]
                          - u[i_jp_k] - u[i_jm_k]
                          - u[i_j_kp] - u[i_j_km]);
}

__global__ void
gpu_difference(const double *u, const double *v, int nx, int ny, int nz)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x ;
	const int j = blockIdx.y * blockDim.y + threadIdx.y ;
	const int k = blockIdx.z * blockDim.z + threadIdx.z ;

	int i_j_k  = i + j*nx + k*nx*ny;

    double du = 0.0;

	if (i>0 && i<nx && j>0 && j<ny && k>0 && k<nz)
       du = fabs(v[i_j_k] - u[i_j_k]);
}


bool GPUScheme::iteration()
{

   gpu_iteration<<<m_GPU->dimBlock, m_GPU->dimGrid>>>(m_u.data(), m_v.data(),
		                                          m_lambda, m_n[0], m_n[1], m_n[2]);
   gpu_difference<<<m_GPU->dimBlock, m_GPU->dimGrid>>>(m_u.data(), m_v.data(),
		                                          m_n[0], m_n[1], m_n[2]);
   return true;
}

