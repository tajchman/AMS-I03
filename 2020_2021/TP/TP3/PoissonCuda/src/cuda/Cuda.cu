#include "Cuda.hxx"
#include "cuda_check.cuh"
#include "timer_id.hxx"

double * allocate(int n) {
    double *d;
    CUDA_CHECK_OP(hipMalloc(&d, n*sizeof(double)));
    return d;
  }
  
  void deallocate(double *&d) {
    Timer & T = GetTimer(T_CopyId); T.start();
    CUDA_CHECK_OP(hipFree(d));
    d = NULL;
    T.stop();
  }
  
  void copyDeviceToHost(double *h, double *d, int n)
  {
    Timer & T = GetTimer(T_CopyId); T.start();
    hipMemcpy(h, d, n * sizeof(double), hipMemcpyDeviceToHost);
    T.stop();
  }
  
  void copyHostToDevice(double *h, double *d, int n)
  {
    Timer & T = GetTimer(T_CopyId); T.start();
    hipMemcpy(h, d, n * sizeof(double), hipMemcpyHostToDevice);
    T.stop();
  }
  
  void copyDeviceToDevice(double *d_out, double *d_in, int n)
  {
    Timer & T = GetTimer(T_CopyId); T.start();
    hipMemcpy(d_out, d_in, n * sizeof(double), hipMemcpyDeviceToDevice);
    T.stop();
  }
  
  
