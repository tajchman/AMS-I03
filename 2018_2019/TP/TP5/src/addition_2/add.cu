#include "hip/hip_runtime.h"
#include "add.hxx"

__global__ void vecAdd(double *c, double *a, double *b, int n)
{
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  
  if (id < n)
    c[id] = a[id] + b[id];
}

void addition_GPU(double **w,
		  double *u,
		  double *v,
		  size_t n)
{
  double *d_u, *d_v, *d_w;
  int blockSize, gridSize;
  
  size_t bytes = n*sizeof(double);
  
  hipMalloc(&d_u, bytes);
  hipMalloc(&d_v, bytes);
  hipMalloc(&d_w, bytes);
  
  hipMemcpy( d_u, u.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy( d_v, v.data(), bytes, hipMemcpyHostToDevice);
 
  blockSize = 1024;
  
  gridSize = (int)ceil((float)n/blockSize);
  
  vecAdd<<<gridSize, blockSize>>>(d_w, d_u, d_v, n);
  
  hipMemcpy(w.data(), d_w, bytes, hipMemcpyDeviceToHost );
  
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
}