#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <stdio.h>
#include <assert.h>
#include "timerGPU.cuh"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.

#include "cuda_check.cuh"

FILE * out_offset, * out_stride;

template <typename T>
__global__ void offset(T* a, int s)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  a[i] = a[i] + 1;
}

template <typename T>
__global__ void stride(T* a, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i] + 1;
}

template <typename T>
void runTest(int deviceId, int nMB)
{
  int blockSize = 256;

  T *d_a;    
  int n = nMB*1024*1024/sizeof(T);

  checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );

  offset<<<n/blockSize, blockSize>>>(d_a, 0); // warm up

  for (int i = 0; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    TimerGPU Tmp;
    Tmp.start();
    
    offset<<<n/blockSize, blockSize>>>(d_a, i);

    Tmp.stop();
    
    fprintf(out_offset, "%d %e\n", i, Tmp.elapsed());
  }

  printf("\n\n");
  stride<<<n/blockSize, blockSize>>>(d_a, 1); // warm up
  for (int i = 1; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    TimerGPU Tmp;
    Tmp.start();
    
    stride<<<n/blockSize, blockSize>>>(d_a, i);

    Tmp.stop();

    fprintf(out_stride, "%d %g\n", i, Tmp.elapsed());
  }

  hipFree(d_a);
}

int main(int argc, char **argv)
{
  int nMB = 4;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {    
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }
  
  hipDeviceProp_t prop;
  
  checkCuda( hipSetDevice(deviceId));
  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", nMB);
  
  printf("%s Precision\n", bFp64 ? "Double" : "Single");

  out_offset = fopen("offset.txt", "w");
  out_stride = fopen("stride.txt", "w");

  if (bFp64) runTest<double>(deviceId, nMB);
  else       runTest<float>(deviceId, nMB);

  fclose(out_offset);
  fclose(out_stride);
}
