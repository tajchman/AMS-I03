#include "hip/hip_runtime.h"
#include <iostream>
#include "calcul.hxx"
#include "timer.hxx"
#include "reduction.h"
#include "cuda_check.cuh"

__global__ void vecInit(double *a, double *b, int n)
{
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  double x;
  
  if (id >= n) return;

  x = 1.0 * id;
  a[id] = sin(x)*sin(x);
  b[id] = cos(x)*cos(x);
}

__global__ void vecAdd(double *c, double *a, double *b, int n)
{
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  
  if (id < n)
    c[id] = a[id] + b[id];
}


Calcul_GPU::Calcul_GPU(std::size_t n) : m_n(n)
{
  Timer T1; T1.start();
  
  std::size_t bytes = m_n*sizeof(double);
  CUDA_CHECK_OP(hipMalloc(&d_u, bytes));
  CUDA_CHECK_OP(hipMalloc(&d_v, bytes));
  CUDA_CHECK_OP(hipMalloc(&d_w, bytes));
  CUDA_CHECK_OP(hipMalloc(&d_tmp, bytes));
    
  T1.stop();
  std::cerr << "\t\ttemps init 1 : " << T1.elapsed() << std::endl;
  Timer T2; T2.start();
  
  blockSize = 512;
  gridSize = (int)ceil((double)n/blockSize);
  
  vecInit<<<gridSize, blockSize>>>(d_u, d_v, n);
  CUDA_CHECK_KERNEL();

  T2.stop();
  std::cerr << "\t\ttemps init 2 : " << T2.elapsed() << std::endl;
}

Calcul_GPU::~Calcul_GPU()
{
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_tmp);
}

void Calcul_GPU::addition()
{
  Timer T; T.start();
  
  vecAdd<<<gridSize, blockSize>>>(d_w, d_u, d_v, m_n);
  CUDA_CHECK_KERNEL();
  
  T.stop();
  std::cerr << "\t\ttemps add.   : " << T.elapsed() << std::endl;
}

double Calcul_GPU::verification()
{
  Timer T; T.start();
  
  double s;
  s = reduce(m_n, d_w, d_tmp, blockSize, gridSize);  
  s = s/m_n - 1.0;
  
  T.stop();
  std::cerr << "\t\ttemps verif. : " << T.elapsed() << std::endl;

  return s;
}

