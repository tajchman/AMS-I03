#include "hip/hip_runtime.h"
#include "add.hxx"

__global__ void vecInit(double *a, double *b, int n)
{
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  double x;
  
  if (id >= n) return;

  x = 1.0 * id;
  a[id] = sin(x)*sin(x);
  b[id] = cos(x)*cos(x);
}

void init_GPU(std::vector<double> &u,
	      std::vector<double> &v)
{
  double *d_u, *d_v;
  int blockSize, gridSize, n = u.size();
  
  size_t bytes = n*sizeof(double);
  
  hipMalloc(&d_u, bytes);
  hipMalloc(&d_v, bytes);
   
  blockSize = 1024;
  
  gridSize = (int)ceil((float)n/blockSize);
  
  vecInit<<<gridSize, blockSize>>>(d_u, d_v, n);
  
  hipMemcpy(u.data(), d_u, bytes, hipMemcpyDeviceToHost );
  hipMemcpy(v.data(), d_v, bytes, hipMemcpyDeviceToHost );
  
  hipFree(d_u);
  hipFree(d_v);
}